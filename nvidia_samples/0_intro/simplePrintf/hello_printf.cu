
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>


__global__ void testKernel(void) {
  printf(" hello world \n ");
}

int main(int argc, char **argv) {
  
  printf("printf() is called. Output:\n\n");

  // Kernel configuration, where a two-dimensional grid and
  // three-dimensional blocks are configured.
  
  testKernel<<<1,1>>>();
  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
